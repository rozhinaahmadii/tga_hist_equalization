#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

using namespace std;

unsigned char *image;
int width, height, pixelWidth;

__global__ void rgb2ycbcr_rowwise(unsigned char* d_image, unsigned int* d_hist, int width, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < height && col < width) {
        int idx = (row * width + col) * 3;
        int r = d_image[idx + 0];
        int g = d_image[idx + 1];
        int b = d_image[idx + 2];

        int Y  = (int)(16 + 0.25679890625 * r + 0.50412890625 * g + 0.09790625 * b);
        int Cb = (int)(128 - 0.168736 * r - 0.331264 * g + 0.5 * b);
        int Cr = (int)(128 + 0.5 * r - 0.418688 * g - 0.081312 * b);

        d_image[idx + 0] = Y;
        d_image[idx + 1] = Cb;
        d_image[idx + 2] = Cr;

        atomicAdd(&(d_hist[Y]), 1);
    }
}

__global__ void equalize_and_reconstruct_rowwise(unsigned char* d_image, int* d_cdf, int width, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < height && col < width) {
        int idx = (row * width + col) * 3;
        int Y  = d_image[idx + 0];
        int Cb = d_image[idx + 1];
        int Cr = d_image[idx + 2];

        int new_Y = d_cdf[Y];

        int R = min(255, max(0, (int)(new_Y + 1.402 * (Cr - 128))));
        int G = min(255, max(0, (int)(new_Y - 0.344136 * (Cb - 128) - 0.714136 * (Cr - 128))));
        int B = min(255, max(0, (int)(new_Y + 1.772 * (Cb - 128))));

        d_image[idx + 0] = R;
        d_image[idx + 1] = G;
        d_image[idx + 2] = B;
    }
}

int eq_GPU(unsigned char* image) {
    int image_size = width * height * pixelWidth;
    unsigned char* d_image;
    unsigned int* d_hist;

    hipMalloc(&d_image, image_size);
    hipMemcpy(d_image, image, image_size, hipMemcpyHostToDevice);

    hipMalloc(&d_hist, 256 * sizeof(unsigned int));
    hipMemset(d_hist, 0, 256 * sizeof(unsigned int));

    // 2D launch configuration
    dim3 block(32, 32);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    // Step 1: RGB → YCbCr and build histogram (GPU)
    rgb2ycbcr_rowwise<<<grid, block>>>(d_image, d_hist, width, height);
    hipDeviceSynchronize();

    // Save the YCbCr image before equalization
    hipMemcpy(image, d_image, image_size, hipMemcpyDeviceToHost);
    stbi_write_png("output_ycbcr_gpu.png", width, height, pixelWidth, image, 0);

    // Copy histogram to CPU
    unsigned int h_hist[256] = {0};
    hipMemcpy(h_hist, d_hist, 256 * sizeof(unsigned int), hipMemcpyDeviceToHost);

    // Compute CDF on CPU
    int h_cdf[256] = {0};
    int sum = 0;
    for (int i = 0; i < 256; i++) {
        sum += h_hist[i];
        h_cdf[i] = (int)((((float)sum - h_hist[0]) / ((float)(width * height - 1))) * 255);
    }

    // Debug range of Y
    int minY = 255, maxY = 0;
    for (int i = 0; i < 256; i++) {
        if (h_hist[i] > 0) {
            minY = min(minY, i);
            maxY = max(maxY, i);
        }
    }
    printf("Y range before equalization: min=%d, max=%d\n", minY, maxY);

    // Copy CDF to GPU
    int* d_cdf;
    hipMalloc(&d_cdf, 256 * sizeof(int));
    hipMemcpy(d_cdf, h_cdf, 256 * sizeof(int), hipMemcpyHostToDevice);

    // Step 2: Apply equalization and convert to RGB
    equalize_and_reconstruct_rowwise<<<grid, block>>>(d_image, d_cdf, width, height);
    hipDeviceSynchronize();

    // Copy final image to CPU
    hipMemcpy(image, d_image, image_size, hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_image);
    hipFree(d_hist);
    hipFree(d_cdf);

    return 0;
}

int main(int argc, char** argv) {
    const char* input = "./IMG/IMG00.jpg";
    const char* output = "output_equalized_gpu.png";

    image = stbi_load(input, &width, &height, &pixelWidth, 0);
    if (!image) {
        fprintf(stderr, "Couldn't load image.\n");
        return -1;
    }

    printf("Loaded image: %s (Width: %d, Height: %d, Channels: %d)\n", input, width, height, pixelWidth);

    struct timeval start, end;
    gettimeofday(&start, NULL);

    eq_GPU(image);

    gettimeofday(&end, NULL);
    long seconds = end.tv_sec - start.tv_sec;
    long micros  = end.tv_usec - start.tv_usec;
    double elapsed_ms = seconds * 1000.0 + micros / 1000.0;

    printf("✅ GPU histogram equalization done in %.3f ms\n", elapsed_ms);

    stbi_write_png(output, width, height, pixelWidth, image, 0);
    stbi_image_free(image);

    return 0;
}
