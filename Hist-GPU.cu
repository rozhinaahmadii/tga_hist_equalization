#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

using namespace std;

unsigned char *image;
int width, height, pixelWidth;

__global__ void rgb2ycbcr_rowwise(unsigned char* d_image, unsigned int* d_hist, int width, int height) {
    int row = blockIdx.x;
    int col = threadIdx.x;

    if (row < height && col < width) {
        int idx = (row * width + col) * 3;
        int r = d_image[idx + 0];
        int g = d_image[idx + 1];
        int b = d_image[idx + 2];

        int Y  = (int)(16 + 0.25679890625 * r + 0.50412890625 * g + 0.09790625 * b);
        int Cb = (int)(128 - 0.168736 * r - 0.331264 * g + 0.5 * b);
        int Cr = (int)(128 + 0.5 * r - 0.418688 * g - 0.081312 * b);

        d_image[idx + 0] = Y;
        d_image[idx + 1] = Cb;
        d_image[idx + 2] = Cr;

        atomicAdd(&(d_hist[Y]), 1);  // now histogram is on GPU
    }
}

__global__ void equalize_and_reconstruct_rowwise(unsigned char* d_image, int* d_cdf, int width, int height) {
    int row = blockIdx.x;
    int col = threadIdx.x;

    if (row < height && col < width) {
        int idx = (row * width + col) * 3;
        int Y  = d_image[idx + 0];
        int Cb = d_image[idx + 1];
        int Cr = d_image[idx + 2];

        int new_Y = d_cdf[Y];

        int R = min(255, max(0, (int)(new_Y + 1.402 * (Cr - 128))));
        int G = min(255, max(0, (int)(new_Y - 0.344136 * (Cb - 128) - 0.714136 * (Cr - 128))));
        int B = min(255, max(0, (int)(new_Y + 1.772 * (Cb - 128))));

        d_image[idx + 0] = R;
        d_image[idx + 1] = G;
        d_image[idx + 2] = B;
    }
}

int eq_GPU(unsigned char* image) {
    int image_size = width * height * pixelWidth;
    unsigned char* d_image;
    unsigned int* d_hist;

    hipMalloc(&d_image, image_size);
    hipMemcpy(d_image, image, image_size, hipMemcpyHostToDevice);

    hipMalloc(&d_hist, 256 * sizeof(unsigned int));
    hipMemset(d_hist, 0, 256 * sizeof(unsigned int));

    dim3 grid(height);
    dim3 block(width);

    // Step 1: RGB → YCbCr and build histogram (row-wise)
    rgb2ycbcr_rowwise<<<grid, block>>>(d_image, d_hist, width, height);
    hipDeviceSynchronize();

    // Copy back the modified YCbCr image to CPU
    hipMemcpy(image, d_image, image_size, hipMemcpyDeviceToHost);
    stbi_write_png("output_original.png", width, height, pixelWidth, image, 0);

    // Copy histogram back to CPU
    unsigned int h_hist[256] = {0};
    hipMemcpy(h_hist, d_hist, 256 * sizeof(unsigned int), hipMemcpyDeviceToHost);

    // Compute CDF on CPU
    int h_cdf[256] = {0};
    int sum = 0;
    for (int i = 0; i < 256; i++) {
        sum += h_hist[i];
        h_cdf[i] = (int)((((float)sum - h_hist[0]) / ((float)(width * height - 1))) * 255);
    }

    // Copy CDF to GPU
    int* d_cdf;
    hipMalloc(&d_cdf, 256 * sizeof(int));
    hipMemcpy(d_cdf, h_cdf, 256 * sizeof(int), hipMemcpyHostToDevice);

    // Step 2: Apply equalization and convert to RGB (row-wise)
    equalize_and_reconstruct_rowwise<<<grid, block>>>(d_image, d_cdf, width, height);
    hipDeviceSynchronize();

    hipMemcpy(image, d_image, image_size, hipMemcpyDeviceToHost);

    hipFree(d_image);
    hipFree(d_hist);
    hipFree(d_cdf);

    return 0;
}

int main(int argc, char** argv) {
    const char* input = "./IMG/IMG00.jpg";
    const char* output = "output_gpu.png";

    image = stbi_load(input, &width, &height, &pixelWidth, 0);
    if (!image) {
        fprintf(stderr, "Couldn't load image.\\n");
        return -1;
    }

    printf("Loaded image: %s (Width: %d, Height: %d, Channels: %d)\\n", input, width, height, pixelWidth);

    struct timeval start, end;
    gettimeofday(&start, NULL);

    eq_GPU(image);

    gettimeofday(&end, NULL);
    long seconds = end.tv_sec - start.tv_sec;
    long micros  = end.tv_usec - start.tv_usec;
    double elapsed_ms = seconds * 1000.0 + micros / 1000.0;

    printf("GPU full row-wise histogram equalization done in %.3f ms\\n", elapsed_ms);

    stbi_write_png(output, width, height, pixelWidth, image, 0);
    stbi_image_free(image);

    return 0;
}
