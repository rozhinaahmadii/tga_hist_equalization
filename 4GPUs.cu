#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

using namespace std;

unsigned char *image;
int width, height, pixelWidth;

// Kernels (sin cambios)
__global__ void rgb2ycbcr_rowwise(unsigned char* d_image, unsigned int* d_hist, int width, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < height && col < width) {
        int idx = (row * width + col) * 3;
        int r = d_image[idx + 0];
        int g = d_image[idx + 1];
        int b = d_image[idx + 2];

        int Y  = (int)(16 + 0.25679890625 * r + 0.50412890625 * g + 0.09790625 * b);
        int Cb = (int)(128 - 0.168736 * r - 0.331264 * g + 0.5 * b);
        int Cr = (int)(128 + 0.5 * r - 0.418688 * g - 0.081312 * b);

        d_image[idx + 0] = Y;
        d_image[idx + 1] = Cb;
        d_image[idx + 2] = Cr;

        atomicAdd(&(d_hist[Y]), 1);
    }
}

__global__ void blur_Y_channel(unsigned char* d_image, unsigned char* d_blurred, int width, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row >= 1 && row < height - 1 && col >= 1 && col < width - 1) {
        float sum = 0.0f;
        for (int dy = -1; dy <= 1; dy++) {
            for (int dx = -1; dx <= 1; dx++) {
                int x = col + dx;
                int y = row + dy;
                int idx = (y * width + x) * 3;
                sum += d_image[idx + 0];
            }
        }
        int out_idx = (row * width + col) * 3;
        d_blurred[out_idx + 0] = (unsigned char)(sum / 9.0f);
        d_blurred[out_idx + 1] = d_image[out_idx + 1];
        d_blurred[out_idx + 2] = d_image[out_idx + 2];
    }
}

__global__ void histogram_shared(unsigned char* d_image, unsigned int* d_hist, int width, int height) {
    __shared__ unsigned int local_hist[256];
    int tid = threadIdx.y * blockDim.x + threadIdx.x;

    for (int i = tid; i < 256; i += blockDim.x * blockDim.y)
        local_hist[i] = 0;

    __syncthreads();

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < height && col < width) {
        int idx = (row * width + col) * 3;
        unsigned char Y = d_image[idx + 0];
        atomicAdd(&local_hist[Y], 1);
    }

    __syncthreads();

    for (int i = tid; i < 256; i += blockDim.x * blockDim.y)
        atomicAdd(&d_hist[i], local_hist[i]);
}

__global__ void equalize_and_reconstruct_rowwise(unsigned char* d_image, int* d_cdf, int width, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < height && col < width) {
        int idx = (row * width + col) * 3;
        int Y  = d_image[idx + 0];
        int Cb = d_image[idx + 1];
        int Cr = d_image[idx + 2];

        int new_Y = d_cdf[Y];

        int R = min(255, max(0, (int)(new_Y + 1.402 * (Cr - 128))));
        int G = min(255, max(0, (int)(new_Y - 0.344136 * (Cb - 128) - 0.714136 * (Cr - 128))));
        int B = min(255, max(0, (int)(new_Y + 1.772 * (Cb - 128))));

        d_image[idx + 0] = R;
        d_image[idx + 1] = G;
        d_image[idx + 2] = B;
    }
}

// Función principal modificada para 4 GPUs
int eq_GPU_multi(unsigned char* image) {
    struct timeval start_total, end_total;
    gettimeofday(&start_total, NULL);

    const int num_gpus = 4;
    
    // Inicializar todas las GPUs
    for (int gpu = 0; gpu < num_gpus; gpu++) {
        hipSetDevice(gpu);
    }

    // Dividir la imagen en 4 partes aproximadamente iguales
    int part_height[num_gpus];
    int remaining = height;
    for (int gpu = 0; gpu < num_gpus; gpu++) {
        part_height[gpu] = (remaining + (num_gpus - gpu - 1)) / (num_gpus - gpu);
        remaining -= part_height[gpu];
    }

    int part_size[num_gpus];
    unsigned char* h_image_part[num_gpus];
    unsigned char* d_image[num_gpus], *d_blurred[num_gpus];
    unsigned int* d_hist[num_gpus];
    int* d_cdf[num_gpus];

    // Calcular tamaños y punteros para cada parte
    h_image_part[0] = image;
    part_size[0] = part_height[0] * width * pixelWidth;
    
    for (int gpu = 1; gpu < num_gpus; gpu++) {
        part_size[gpu] = part_height[gpu] * width * pixelWidth;
        h_image_part[gpu] = h_image_part[gpu-1] + part_size[gpu-1];
    }

    // Asignar memoria en cada GPU
    for (int gpu = 0; gpu < num_gpus; gpu++) {
        hipSetDevice(gpu);
        hipMalloc((void**)&d_image[gpu], part_size[gpu]);
        hipMalloc((void**)&d_blurred[gpu], part_size[gpu]);
        hipMalloc((void**)&d_hist[gpu], 256 * sizeof(unsigned int));
        hipMalloc((void**)&d_cdf[gpu], 256 * sizeof(int));
    }

    // Copiar datos a cada GPU (asíncrono con streams)
    hipStream_t streams[num_gpus];
    for (int gpu = 0; gpu < num_gpus; gpu++) {
        hipSetDevice(gpu);
        hipStreamCreate(&streams[gpu]);
        hipMemcpyAsync(d_image[gpu], h_image_part[gpu], part_size[gpu], 
                       hipMemcpyHostToDevice, streams[gpu]);
    }

    // Configurar kernels
    dim3 block(32, 32);

    // Ejecutar kernels en paralelo en todas las GPUs
    hipEvent_t events[num_gpus];
    for (int gpu = 0; gpu < num_gpus; gpu++) {
        hipSetDevice(gpu);
        hipEventCreate(&events[gpu]);
        
        dim3 grid((width + block.x - 1) / block.x, (part_height[gpu] + block.y - 1) / block.y);

        // Paso 1: RGB → YCbCr + histograma inicial
        rgb2ycbcr_rowwise<<<grid, block, 0, streams[gpu]>>>(d_image[gpu], d_hist[gpu], width, part_height[gpu]);
        
        // Paso 2: Blur Y channel
        blur_Y_channel<<<grid, block, 0, streams[gpu]>>>(d_image[gpu], d_blurred[gpu], width, part_height[gpu]);
        hipMemcpyAsync(d_image[gpu], d_blurred[gpu], part_size[gpu], 
                       hipMemcpyDeviceToDevice, streams[gpu]);
        
        // Paso 3: Histograma con memoria compartida
        hipMemsetAsync(d_hist[gpu], 0, 256 * sizeof(unsigned int), streams[gpu]);
        histogram_shared<<<grid, block, 0, streams[gpu]>>>(d_image[gpu], d_hist[gpu], width, part_height[gpu]);
        
        hipEventRecord(events[gpu], streams[gpu]);
    }

    // Esperar que todas las GPUs terminen
    for (int gpu = 0; gpu < num_gpus; gpu++) {
        hipSetDevice(gpu);
        hipEventSynchronize(events[gpu]);
    }

    // Combinar histogramas parciales en CPU
    unsigned int h_hist_combined[256] = {0};
    for (int gpu = 0; gpu < num_gpus; gpu++) {
        unsigned int h_hist_partial[256];
        hipSetDevice(gpu);
        hipMemcpy(h_hist_partial, d_hist[gpu], 256 * sizeof(unsigned int), hipMemcpyDeviceToHost);
        for (int i = 0; i < 256; i++) h_hist_combined[i] += h_hist_partial[i];
    }

    // Calcular CDF global
    int h_cdf[256] = {0}, sum = 0;
    for (int i = 0; i < 256; i++) {
        sum += h_hist_combined[i];
        h_cdf[i] = (int)(((float)sum - h_hist_combined[0]) / (width * height - 1) * 255);
    }

    // Copiar CDF a todas las GPUs
    for (int gpu = 0; gpu < num_gpus; gpu++) {
        hipSetDevice(gpu);
        hipMemcpy(d_cdf[gpu], h_cdf, 256 * sizeof(int), hipMemcpyHostToDevice);
    }

    // Aplicar equalización y reconstrucción
    for (int gpu = 0; gpu < num_gpus; gpu++) {
        hipSetDevice(gpu);
        dim3 grid((width + block.x - 1) / block.x, (part_height[gpu] + block.y - 1) / block.y);
        equalize_and_reconstruct_rowwise<<<grid, block, 0, streams[gpu]>>>(d_image[gpu], d_cdf[gpu], width, part_height[gpu]);
        hipMemcpyAsync(h_image_part[gpu], d_image[gpu], part_size[gpu], 
                       hipMemcpyDeviceToHost, streams[gpu]);
    }

    // Sincronizar todas las GPUs
    for (int gpu = 0; gpu < num_gpus; gpu++) {
        hipSetDevice(gpu);
        hipStreamSynchronize(streams[gpu]);
    }

    // Liberar memoria
    for (int gpu = 0; gpu < num_gpus; gpu++) {
        hipSetDevice(gpu);
        hipFree(d_image[gpu]);
        hipFree(d_blurred[gpu]);
        hipFree(d_hist[gpu]);
        hipFree(d_cdf[gpu]);
        hipStreamDestroy(streams[gpu]);
        hipEventDestroy(events[gpu]);
    }

    gettimeofday(&end_total, NULL);
    double total_time = (end_total.tv_sec - start_total.tv_sec) * 1000.0 + 
                       (end_total.tv_usec - start_total.tv_usec) / 1000.0;
    printf("\n✅ Total GPU (%d GPUs) time: %.3f ms\n", num_gpus, total_time);

    return 0;
}

// Función main (sin cambios necesarios)
int main(int argc, char** argv) {
    const char* input = "./IMG/IMG00.jpg";
    const char* output = "output_equalized_4gpu.png";

    int n_channels;
    unsigned char* raw = stbi_load(input, &width, &height, &n_channels, 0);
    if (!raw) {
        fprintf(stderr, "❌ Couldn't load image.\n");
        return -1;
    }

    pixelWidth = n_channels;
    int size = width * height * pixelWidth;

    // Allocate pinned memory
    hipHostAlloc((void**)&image, size, hipHostMallocDefault);
    memcpy(image, raw, size);
    stbi_image_free(raw);

    printf("📷 Image loaded: %d x %d (Channels: %d)\n", width, height, pixelWidth);

    // Verificar número de GPUs disponibles
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    printf("Number of CUDA devices: %d\n", deviceCount);

    // Ejecutar en 4 GPUs
    struct timeval start, end;
    gettimeofday(&start, NULL);
    eq_GPU_multi(image);
    gettimeofday(&end, NULL);

    double elapsed = (end.tv_sec - start.tv_sec) * 1000.0 + 
                    (end.tv_usec - start.tv_usec) / 1000.0;
    printf("🕒 Total runtime (incl. GPU): %.3f ms\n", elapsed);

    // Guardar imagen
    stbi_write_png(output, width, height, pixelWidth, image, 0);
    hipHostFree(image);

    return 0;
}
