#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

using namespace std;

unsigned char *image;
int width, height, pixelWidth;

__global__ void rgb2ycbcr_rowwise(unsigned char* d_image, int width, int height) {
    int row = blockIdx.x;
    int col = threadIdx.x;

    if (row < height && col < width) {
        int idx = (row * width + col) * 3;
        int r = d_image[idx + 0];
        int g = d_image[idx + 1];
        int b = d_image[idx + 2];

        int Y  = (int)(16 + 0.25679890625 * r + 0.50412890625 * g + 0.09790625 * b);
        int Cb = (int)(128 - 0.168736 * r - 0.331264 * g + 0.5 * b);
        int Cr = (int)(128 + 0.5 * r - 0.418688 * g - 0.081312 * b);

        d_image[idx + 0] = Y;
        d_image[idx + 1] = Cb;
        d_image[idx + 2] = Cr;
    }
}

__global__ void equalize_and_reconstruct_rowwise(unsigned char* d_image, int* d_cdf, int width, int height) {
    int row = blockIdx.x;
    int col = threadIdx.x;

    if (row < height && col < width) {
        int idx = (row * width + col) * 3;
        int Y  = d_image[idx + 0];
        int Cb = d_image[idx + 1];
        int Cr = d_image[idx + 2];

        int new_Y = d_cdf[Y];

        int R = min(255, max(0, (int)(new_Y + 1.402 * (Cr - 128))));
        int G = min(255, max(0, (int)(new_Y - 0.344136 * (Cb - 128) - 0.714136 * (Cr - 128))));
        int B = min(255, max(0, (int)(new_Y + 1.772 * (Cb - 128))));

        d_image[idx + 0] = R;
        d_image[idx + 1] = G;
        d_image[idx + 2] = B;
    }
}

int eq_GPU(unsigned char* image) {
    int image_size = width * height * pixelWidth;
    unsigned char* d_image;

    hipMalloc(&d_image, image_size);
    hipMemcpy(d_image, image, image_size, hipMemcpyHostToDevice);

    dim3 grid(height);
    dim3 block(width);

    // Step 1: RGB → YCbCr (row-wise)
    rgb2ycbcr_rowwise<<<grid, block>>>(d_image, width, height);
    hipDeviceSynchronize();

    // Save intermediate original (YCbCr) image
    hipMemcpy(image, d_image, image_size, hipMemcpyDeviceToHost);
    stbi_write_png("output_original.png", width, height, pixelWidth, image, 0);

    // Step 2: Histogram + CDF on CPU
    int histogram[256] = {0};
    for (int i = 0; i < width * height * 3; i += 3) {
        int Y = image[i];
        histogram[Y]++;
    }

    int cdf[256] = {0};
    int sum = 0;
    for (int i = 0; i < 256; i++) {
        sum += histogram[i];
        cdf[i] = (int)((((float)sum - histogram[0]) / ((float)(width * height - 1))) * 255);
    }

    // Copy back the modified YCbCr image
    hipMemcpy(d_image, image, image_size, hipMemcpyHostToDevice);

    // Copy CDF to GPU
    int* d_cdf;
    hipMalloc(&d_cdf, 256 * sizeof(int));
    hipMemcpy(d_cdf, cdf, 256 * sizeof(int), hipMemcpyHostToDevice);

    // Step 3: Equalize and reconstruct RGB (row-wise)
    equalize_and_reconstruct_rowwise<<<grid, block>>>(d_image, d_cdf, width, height);
    hipDeviceSynchronize();

    hipMemcpy(image, d_image, image_size, hipMemcpyDeviceToHost);
    hipFree(d_image);
    hipFree(d_cdf);

    return 0;
}

int main(int argc, char** argv) {
    const char* input = "./IMG/IMG00.jpg";
    const char* output = "output_gpu.png";

    image = stbi_load(input, &width, &height, &pixelWidth, 0);
    if (!image) {
        fprintf(stderr, "Couldn't load image.\\n");
        return -1;
    }

    printf("Loaded image: %s (Width: %d, Height: %d, Channels: %d)\\n", input, width, height, pixelWidth);

    struct timeval start, end;
    gettimeofday(&start, NULL);

    eq_GPU(image);

    gettimeofday(&end, NULL);
    long seconds = end.tv_sec - start.tv_sec;
    long micros  = end.tv_usec - start.tv_usec;
    double elapsed_ms = seconds * 1000.0 + micros / 1000.0;

    printf(" GPU histogram equalization done in %.3f ms\\n", elapsed_ms);
    stbi_write_png(output, width, height, pixelWidth, image, 0);
    stbi_image_free(image);

    return 0;
}
